
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <math.h>
#include <unistd.h>
#include <semaphore.h>
#include <hip/hip_runtime_api.h>

/******************************************************************************
 * This program takes an initial estimate of m and c and finds the associated 
 * rms error. It is then as a base to generate and evaluate 8 new estimates, 
 * which are steps in different directions in m-c space. The best estimate is 
 * then used as the base for another iteration of "generate and evaluate". This 
 * continues until none of the new estimates are better than the base. This is
 * a gradient search for a minimum in mc-space.
 * 
 * To compile:
 *   nvcc -o lr_coursework lr_coursework.c -lm
 * 
 * To run:
 *   ./lr_coursework
 * 
 * Dr Kevan Buckley, University of Wolverhampton, 2018
 *****************************************************************************/



typedef struct point_t {
	double x;
	double y;
} point_t;

typedef struct mean_intercept 
{
	double mean,intercept,error;
} mean_intercept;

int n_data = 1000;
point_t data[] = {
  {83.40,147.61},{72.54,104.92},{65.45,133.77},{73.47,125.99},
  {76.86,154.92},{65.45,128.87},{78.30,144.23},{65.46,125.35},
  {90.55,155.58},{34.15,79.11},{80.14,148.12},{49.14,88.19},
  {29.59,71.28},{33.80,87.85},{60.49,113.09},{61.86,104.64},
  {51.46,103.14},{92.22,146.63},{20.85,75.45},{37.35,85.67},
  {37.03,81.01},{78.28,129.70},{24.30,72.23},{51.80,71.08},
  { 7.25,51.68},{23.35,81.55},{ 4.15,34.78},{17.35,65.95},
  {88.79,138.66},{14.42,44.89},{ 9.99,37.55},{21.65,61.09},
  {93.37,156.67},{93.84,158.24},{17.64,35.28},{88.45,143.46},
  {18.73,59.89},{25.84,62.90},{ 0.57,22.82},{45.79,95.26},
  {35.82,56.89},{87.20,158.45},{21.71,51.77},{87.34,145.72},
  {86.74,146.13},{39.91,84.15},{78.96,137.15},{85.24,158.70},
  {66.42,113.25},{75.63,127.04},{74.44,143.61},{23.77,67.40},
  {20.08,53.86},{48.10,86.28},{16.65,71.25},{41.24,85.45},
  {52.07,120.57},{46.09,88.49},{89.30,164.35},{10.87,43.98},
  {68.84,118.22},{71.18,128.73},{63.66,118.07},{ 8.91,49.19},
  {88.47,168.07},{67.78,108.50},{53.59,106.40},{53.50,92.92},
  {83.77,144.06},{19.14,61.12},{48.93,77.52},{88.98,137.77},
  {73.03,142.85},{18.12,51.13},{47.41,114.19},{85.65,154.59},
  { 8.95,29.40},{30.96,61.29},{11.83,48.41},{40.23,87.39},
  {61.20,126.26},{87.31,132.01},{48.23,96.56},{73.60,143.08},
  {50.48,113.60},{32.02,73.81},{19.67,46.99},{37.55,76.24},
  { 3.33,31.40},{32.43,49.02},{40.21,83.78},{17.80,59.31},
  { 5.22,28.61},{88.87,159.96},{46.16,90.25},{ 2.36,46.28},
  {51.44,90.08},{93.40,142.52},{ 4.98,49.09},{36.76,97.32},
  {23.77,49.73},{89.97,134.98},{ 8.82,50.94},{50.48,93.56},
  {65.04,114.55},{60.55,111.27},{35.97,73.02},{74.96,122.72},
  {12.86,40.67},{97.12,143.67},{89.34,166.30},{15.32,63.80},
  {28.54,72.89},{32.18,57.93},{44.06,104.11},{65.79,121.85},
  {98.20,158.24},{ 7.18,44.61},{58.37,120.42},{33.07,69.07},
  {42.11,93.45},{67.78,124.10},{93.68,178.02},{92.53,150.69},
  { 4.23,45.46},{84.16,155.47},{83.87,129.94},{96.19,162.44},
  {53.78,108.12},{60.75,111.66},{42.86,97.22},{59.00,125.00},
  {68.69,108.10},{33.26,65.97},{89.96,159.07},{59.56,111.42},
  {38.44,77.52},{84.89,128.41},{53.71,97.76},{84.69,147.55},
  {34.92,80.25},{21.00,32.76},{24.51,58.35},{35.35,63.64},
  { 5.72,38.76},{40.64,99.03},{47.22,89.87},{31.01,69.53},
  {13.80,44.85},{47.94,86.88},{95.28,167.60},{52.76,113.81},
  {96.62,154.41},{87.37,150.13},{33.17,78.65},{22.95,57.40},
  {45.54,81.41},{80.68,154.77},{54.78,101.60},{28.94,65.96},
  {40.51,88.07},{65.97,130.50},{48.89,97.36},{ 8.97,40.79},
  {87.46,150.28},{ 7.08,58.26},{11.21,40.91},{33.04,77.84},
  {76.77,133.75},{32.99,71.45},{ 1.25,29.89},{13.05,57.13},
  {31.55,68.39},{20.69,63.25},{74.36,124.05},{87.83,169.17},
  {37.17,74.95},{21.69,69.15},{77.26,140.43},{56.52,96.59},
  {78.25,124.38},{ 9.80,48.77},{60.95,126.66},{46.63,86.70},
  {94.97,161.81},{61.19,133.42},{85.30,157.05},{88.37,150.54},
  {30.38,80.82},{40.73,86.27},{48.27,93.38},{46.61,118.92},
  {51.96,112.30},{82.10,150.30},{58.30,102.70},{28.97,82.34},
  {81.84,156.07},{85.89,142.69},{90.17,160.60},{58.45,103.11},
  {79.40,137.68},{94.20,159.11},{66.26,121.98},{44.45,101.81},
  {22.52,54.95},{20.53,54.93},{97.16,138.84},{29.85,82.22},
  {58.75,95.56},{ 0.99,31.15},{35.24,79.11},{10.50,32.69},
  {45.92,80.33},{33.26,61.15},{61.26,125.23},{90.61,151.24},
  {29.02,77.77},{18.43,67.08},{63.61,123.62},{ 4.95,54.72},
  {91.58,164.86},{ 2.86,26.95},{10.89,34.21},{69.77,129.93},
  {20.43,56.17},{91.07,151.94},{ 8.72,55.49},{60.14,101.94},
  {50.19,110.14},{37.79,92.15},{87.60,153.34},{80.46,140.40},
  {10.83,45.06},{19.50,47.35},{34.86,71.46},{82.55,140.81},
  {36.35,92.74},{31.85,75.31},{98.21,168.27},{25.65,64.68},
  { 4.79,46.84},{18.89,50.58},{89.52,160.34},{61.00,102.25},
  {62.49,101.47},{66.21,120.08},{32.70,66.67},{85.58,147.20},
  {59.00,116.78},{19.79,64.14},{ 2.64,33.66},{55.75,112.67},
  {80.93,147.05},{49.55,91.02},{47.86,86.08},{61.20,101.88},
  {42.73,75.88},{15.85,51.98},{56.58,100.71},{65.74,120.30},
  {89.83,139.14},{23.74,58.40},{66.65,121.82},{75.30,127.15},
  {81.00,129.52},{78.99,142.73},{43.96,71.36},{93.42,157.45},
  {54.27,98.74},{ 9.46,46.38},{12.22,51.82},{96.14,156.01},
  {28.15,69.90},{47.06,102.39},{65.93,124.04},{23.25,66.72},
  {27.46,70.99},{19.40,52.03},{40.86,78.16},{11.91,49.30},
  {81.71,149.06},{84.59,132.85},{99.69,156.53},{45.76,100.75},
  {43.89,103.43},{ 5.46,61.86},{68.30,128.61},{85.41,154.66},
  {93.96,157.56},{ 0.54,38.56},{60.89,99.50},{99.57,166.06},
  {12.63,50.08},{57.83,101.57},{44.80,81.37},{ 3.66,50.59},
  {10.78,21.86},{ 9.47,49.11},{32.12,74.24},{84.46,135.50},
  {82.40,133.58},{34.94,74.69},{37.31,87.09},{ 0.98,29.20},
  {35.72,71.97},{41.90,95.63},{34.26,71.61},{64.62,123.18},
  {51.52,96.68},{26.67,72.74},{53.44,107.30},{42.70,87.66},
  {41.63,92.62},{10.04,44.58},{11.52,52.76},{27.46,70.45},
  {52.10,108.50},{13.78,66.70},{83.67,150.71},{83.01,152.01},
  {12.59,56.94},{ 8.03,50.32},{ 1.82,45.23},{34.37,57.19},
  {11.47,31.80},{92.91,126.10},{ 2.40,36.52},{62.51,116.81},
  {46.88,105.13},{53.38,110.69},{83.84,138.87},{91.99,167.77},
  {71.92,124.48},{39.36,84.07},{48.84,100.12},{99.64,169.86},
  {37.00,93.86},{31.98,82.30},{87.31,142.57},{45.93,93.09},
  { 6.42,48.32},{75.90,146.51},{ 2.14,23.27},{ 6.73,51.36},
  {48.43,92.35},{52.32,97.82},{57.85,107.39},{39.53,79.74},
  {69.47,117.61},{23.80,62.52},{ 2.79,29.28},{ 4.22,49.86},
  {76.98,144.11},{43.84,87.16},{12.92,40.18},{39.93,87.48},
  {75.42,118.12},{39.90,86.14},{52.81,114.23},{83.17,146.47},
  { 9.68,48.13},{69.79,122.59},{15.47,53.40},{39.36,94.16},
  {42.72,107.97},{58.18,93.58},{30.34,76.23},{ 4.26,24.92},
  {26.24,73.74},{53.53,107.66},{29.63,68.98},{59.69,110.63},
  {46.12,95.88},{65.15,113.95},{94.83,144.88},{61.43,101.46},
  {79.21,135.09},{88.66,155.32},{51.55,101.32},{41.81,98.30},
  {96.05,161.13},{44.75,108.04},{22.12,66.55},{24.89,62.24},
  {31.15,86.25},{36.86,91.64},{ 7.99,56.86},{22.93,63.10},
  {64.98,90.77},{58.74,125.70},{20.54,55.25},{78.33,137.23},
  {82.73,153.66},{11.39,46.82},{19.32,61.82},{26.50,71.45},
  { 7.50,49.88},{65.94,126.63},{35.42,72.80},{76.44,141.29},
  { 6.09,60.21},{52.65,117.56},{52.39,101.29},{25.83,70.15},
  {33.56,69.65},{ 7.33,37.91},{11.41,42.83},{56.62,112.19},
  { 6.50,41.54},{65.36,115.60},{86.39,132.14},{ 6.46,22.42},
  {53.02,116.69},{11.39,42.43},{49.49,102.72},{58.35,105.64},
  {48.49,93.73},{53.84,96.47},{44.67,83.11},{12.52,54.13},
  {81.10,154.15},{ 8.91,55.18},{55.47,108.23},{59.27,125.01},
  {40.15,105.41},{62.31,128.10},{ 2.64,31.22},{91.46,153.05},
  {74.79,137.07},{22.00,60.85},{48.25,66.27},{31.07,55.35},
  {99.43,167.69},{58.61,110.83},{ 8.74,45.26},{40.89,81.83},
  { 8.07,46.47},{81.47,143.52},{20.48,62.89},{66.21,121.29},
  {64.47,131.27},{23.27,75.31},{25.81,97.23},{81.82,141.75},
  {57.93,102.03},{32.67,80.39},{53.58,115.67},{73.34,141.86},
  {98.22,159.46},{17.55,57.75},{ 5.13,53.34},{40.18,87.19},
  {78.34,132.14},{71.08,136.72},{74.29,128.71},{52.15,110.47},
  {33.71,81.08},{51.33,87.46},{29.77,94.79},{28.26,76.34},
  {92.57,157.40},{93.84,172.74},{ 9.13,51.03},{23.50,46.37},
  {57.44,90.30},{10.05,50.90},{ 8.47,33.74},{11.35,59.42},
  {78.53,123.18},{97.12,164.55},{83.59,134.47},{55.47,118.31},
  {38.25,80.18},{21.33,62.64},{27.82,83.56},{32.73,55.80},
  {71.17,133.84},{92.01,157.99},{17.62,48.16},{82.54,158.45},
  {40.62,77.54},{43.98,85.94},{66.45,136.31},{66.20,119.80},
  {30.71,70.20},{93.78,152.38},{88.71,154.33},{28.83,74.37},
  {64.97,125.14},{64.85,111.34},{70.43,122.25},{77.78,123.19},
  {45.93,100.59},{13.38,43.19},{52.62,96.97},{83.78,142.15},
  {42.80,79.58},{67.94,106.57},{24.08,61.09},{75.76,125.71},
  { 4.52,42.12},{89.80,148.31},{ 7.62,39.74},{26.90,66.35},
  {60.34,124.21},{83.29,138.70},{52.39,104.57},{55.97,112.73},
  {78.80,129.25},{17.03,47.69},{58.27,109.93},{48.99,101.15},
  {58.57,100.57},{51.17,80.95},{20.86,80.08},{69.26,120.07},
  { 5.56,34.11},{56.44,111.58},{56.38,112.28},{25.47,92.76},
  {71.30,141.49},{26.30,66.17},{91.63,153.64},{86.31,150.87},
  {71.08,127.56},{18.72,69.86},{46.69,77.36},{29.02,69.91},
  {64.62,111.50},{62.67,129.28},{30.06,74.08},{53.66,93.78},
  {90.54,138.90},{23.41,72.52},{50.30,75.92},{21.96,51.95},
  {50.39,90.31},{10.12,47.62},{38.51,71.56},{80.32,161.92},
  {67.59,134.83},{32.94,69.58},{50.68,110.03},{55.14,102.49},
  {35.53,73.83},{27.85,71.35},{37.87,95.68},{77.13,134.39},
  {62.66,96.90},{38.97,90.73},{11.39,39.76},{41.97,90.25},
  {48.25,98.17},{78.93,139.09},{29.31,87.45},{30.63,65.73},
  {84.20,141.00},{89.57,165.60},{ 9.71,44.10},{23.07,51.55},
  {54.70,92.49},{92.63,147.99},{39.05,77.61},{30.13,77.69},
  {96.19,164.77},{35.73,88.90},{62.12,119.52},{94.80,162.03},
  {81.35,141.94},{ 0.03,30.20},{76.16,140.54},{26.10,86.91},
  {75.44,137.38},{97.34,166.59},{24.75,86.17},{96.95,169.17},
  {37.96,91.41},{59.64,94.86},{80.90,137.20},{62.06,127.69},
  {49.15,81.28},{66.99,131.85},{27.80,85.89},{94.81,155.85},
  {70.15,124.55},{40.24,99.85},{75.97,140.63},{62.89,111.40},
  {97.96,161.36},{29.10,74.01},{86.77,155.58},{ 1.72,45.19},
  {84.14,146.98},{53.53,101.31},{44.61,86.68},{88.78,145.86},
  {89.13,152.13},{43.27,84.02},{21.45,62.22},{39.51,95.07},
  {60.87,111.71},{32.46,98.03},{42.22,88.28},{11.35,51.14},
  {65.75,103.46},{97.34,165.81},{ 1.95,34.25},{34.67,54.13},
  {91.65,130.56},{52.66,83.06},{ 5.28,43.97},{16.27,38.54},
  {45.90,91.47},{98.75,173.56},{38.59,83.02},{30.95,74.27},
  {52.35,89.28},{15.20,45.19},{78.63,138.11},{68.61,112.98},
  {27.40,83.10},{37.56,93.54},{73.93,119.74},{78.76,126.99},
  {19.34,64.20},{41.68,88.98},{46.86,104.98},{64.65,126.49},
  { 9.84,42.12},{82.19,158.11},{84.25,164.14},{66.91,128.93},
  {18.23,51.05},{30.16,51.36},{ 3.22,49.29},{56.16,102.79},
  {73.30,127.84},{ 9.61,46.51},{69.38,126.25},{61.21,119.13},
  { 6.80,56.52},{45.93,85.27},{65.19,126.47},{35.62,67.20},
  {75.92,127.33},{85.11,154.35},{38.06,80.23},{83.37,133.27},
  {38.99,104.31},{49.04,98.81},{22.22,69.14},{92.11,153.89},
  {64.47,117.69},{73.09,123.90},{89.23,141.91},{89.64,149.03},
  {37.18,78.53},{ 8.09,48.76},{29.75,73.52},{65.93,115.28},
  {11.59,51.08},{56.33,121.44},{17.43,44.22},{81.80,126.95},
  {10.48,42.98},{ 9.56,51.95},{57.53,109.77},{39.33,104.40},
  {72.87,126.32},{ 5.48,27.14},{ 5.55,35.27},{16.47,59.61},
  {26.20,77.42},{22.54,55.07},{34.68,95.92},{29.58,60.17},
  {85.86,139.31},{97.99,159.54},{89.28,159.18},{35.95,93.54},
  { 9.30,52.11},{69.28,138.57},{66.97,135.08},{33.89,64.49},
  { 3.44,33.11},{73.07,128.74},{80.15,117.34},{73.89,120.38},
  {33.96,78.74},{51.70,93.38},{22.73,59.00},{41.65,86.22},
  {62.65,125.59},{32.89,67.71},{97.63,171.24},{88.86,164.21},
  {41.43,92.68},{ 0.45,38.49},{15.81,58.56},{15.13,44.16},
  {37.25,86.96},{43.15,88.16},{ 4.28,30.20},{88.71,161.50},
  {38.96,77.97},{90.28,145.13},{64.52,101.31},{15.35,57.64},
  {49.62,93.49},{ 3.52,25.43},{82.41,141.06},{91.46,168.64},
  {82.11,141.20},{87.83,145.49},{75.96,115.41},{89.78,150.54},
  {52.64,88.18},{20.56,56.77},{20.57,53.84},{ 0.25,27.55},
  {95.28,155.47},{44.27,99.18},{95.02,145.89},{ 1.98,45.53},
  {93.87,143.86},{14.10,56.01},{38.17,75.59},{19.86,58.24},
  {45.36,101.90},{ 6.95,63.58},{85.82,148.00},{68.90,118.55},
  {22.57,69.75},{47.92,111.17},{79.53,147.81},{91.50,147.24},
  {57.42,109.40},{34.40,91.33},{96.98,142.69},{73.21,141.54},
  {51.26,106.86},{95.83,171.12},{28.48,69.67},{67.52,131.14},
  {41.38,102.69},{54.51,107.24},{16.92,65.82},{36.83,87.52},
  {89.65,160.06},{20.76,63.42},{26.09,63.15},{ 7.32,36.77},
  {21.28,54.99},{37.48,88.59},{ 7.49,39.10},{22.38,64.85},
  {47.68,95.43},{99.11,162.36},{68.55,126.50},{ 1.37,51.29},
  {21.32,68.06},{26.85,68.71},{92.79,160.61},{77.57,133.32},
  {54.96,91.78},{41.13,92.66},{97.83,158.15},{17.50,58.32},
  {74.84,130.28},{81.59,141.48},{59.88,111.57},{58.20,98.21},
  {74.47,142.77},{58.72,118.42},{45.35,87.70},{38.49,92.27},
  {78.76,120.57},{12.91,55.02},{55.94,112.05},{52.81,83.76},
  {65.45,137.16},{62.12,114.92},{19.86,63.73},{39.40,104.00},
  {87.23,141.21},{54.12,101.50},{55.56,122.21},{17.95,44.35},
  {62.78,111.00},{10.52,37.86},{91.96,153.33},{42.66,89.54},
  {11.47,75.24},{96.49,165.44},{43.89,90.74},{20.36,51.37},
  {28.80,65.54},{45.55,104.84},{16.73,61.97},{82.40,144.38},
  {86.32,144.22},{70.70,113.32},{92.80,143.45},{39.04,90.76},
  {71.05,130.11},{26.39,59.69},{97.09,161.46},{80.81,147.79},
  {42.44,79.70},{41.30,85.83},{39.15,94.91},{55.08,117.42},
  { 4.29,51.07},{82.12,147.32},{65.83,112.89},{68.59,117.95},
  {15.10,35.93},{81.56,134.13},{ 8.34,50.35},{76.84,137.93},
  {61.72,115.31},{88.78,132.13},{19.61,64.58},{96.26,151.97},
  { 8.70,32.30},{ 7.61,28.60},{ 3.62,41.73},{97.46,158.55},
  {47.46,96.34},{77.40,124.96},{71.70,138.33},{12.50,43.39},
  {84.76,131.33},{76.32,150.87},{53.61,105.47},{91.55,156.41},
  {45.13,75.78},{52.21,102.83},{83.74,152.60},{13.47,48.42},
  {55.23,91.84},{61.12,115.02},{84.30,133.30},{15.93,60.33},
  {83.73,142.73},{92.37,139.03},{ 3.68,41.01},{71.95,127.08},
  {54.27,114.72},{52.37,107.05},{72.98,134.09},{10.23,48.74},
  { 5.08,44.39},{89.88,155.83},{24.53,65.76},{17.75,46.44},
  {47.44,74.55},{67.34,108.83},{85.39,151.12},{28.11,69.97},
  {58.66,135.29},{50.25,99.45},{79.97,138.24},{83.07,133.74},
  {92.45,156.72},{75.52,137.73},{58.28,125.05},{27.89,92.72},
  {99.19,165.98},{85.69,150.24},{63.90,123.00},{73.80,129.28},
  {24.32,74.44},{30.39,82.69},{60.82,108.06},{26.45,81.04},
  {37.23,97.70},{ 2.88,45.20},{ 1.83,44.48},{33.11,66.83},
  {50.58,101.71},{52.33,96.31},{72.32,126.15},{16.40,38.44},
  {53.81,121.79},{15.00,68.52},{24.89,61.86},{88.21,157.81},
  {71.94,134.93},{27.44,68.86},{31.65,67.28},{ 7.26,66.51},
  {75.74,134.12},{22.47,58.44},{32.61,92.25},{28.87,74.50},
  {32.92,77.74},{97.41,148.42},{16.19,59.59},{24.47,50.23},
  {43.14,74.40},{40.40,86.38},{56.27,108.81},{14.47,47.54},
  {90.07,158.11},{82.14,146.16},{83.47,144.99},{62.35,121.20},
  {42.20,82.19},{37.64,91.81},{69.08,88.79},{32.14,80.95},
  { 5.52,40.14},{55.71,89.94},{26.05,62.69},{49.56,100.16},
  {79.13,142.95},{34.78,68.33},{42.80,89.66},{27.22,40.42},
  {60.15,104.90},{91.22,161.89},{ 3.04,59.14},{95.86,177.76},
  {98.69,160.57},{61.47,107.57},{67.00,146.75},{38.29,65.52},
  {19.73,64.24},{20.96,68.66},{25.99,69.36},{68.08,116.16},
  {17.99,59.41},{44.36,95.32},{24.19,54.94},{40.07,96.17},
  {64.16,113.89},{ 6.80,52.64},{62.71,102.79},{70.46,114.76},
  {75.60,133.55},{32.76,71.87},{19.13,60.08},{49.90,92.52},
  {35.31,72.27},{61.72,133.80},{86.50,145.53},{11.83,45.72},
  {20.54,67.86},{44.27,119.83},{88.69,169.41},{50.40,101.31},
  {41.16,81.55},{49.37,99.91},{28.17,74.30},{ 0.67,32.54},
  {75.04,125.67},{43.73,98.11},{98.29,161.62},{ 2.78,37.29},
  { 2.87,35.28},{65.57,98.16},{55.35,121.82},{76.59,114.64},
  {28.37,73.69},{49.68,79.61},{77.90,125.07},{68.58,134.97},
  {36.81,78.69},{47.34,93.22},{55.76,122.80},{77.30,140.19},
  {69.24,118.57},{ 6.26,43.91},{31.13,69.37},{25.27,40.97},
  {51.55,87.23},{72.00,129.52},{67.84,117.33},{16.75,61.75},
  {60.46,123.82},{86.48,144.19},{32.65,89.43},{19.37,64.02},
  {43.58,77.28},{39.18,71.92},{12.25,47.77},{55.90,121.56},
  {91.01,145.92},{57.97,95.12},{ 0.72,39.01},{50.36,97.80},
  {83.65,145.58},{28.05,63.44},{70.27,125.92},{64.64,97.51},
  {42.99,94.15},{51.87,118.66},{19.72,65.03},{88.40,140.06}
};


double residual_error(double x, double y, double m, double c) {
	double e = (m * x) + c - y;
  	return e * e;
}

__device__ double residual_err(double x, double y, double m, double c) {
	double e = (m * x) + c - y;
  	return e * e;
}



int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference) {
  	long long int ds =  finish->tv_sec - start->tv_sec; 
  	long long int dn =  finish->tv_nsec - start->tv_nsec; 

	if(dn < 0 ) {
    	ds--;
    	dn += 1000000000; 
  	} 
  	*difference = ds * 1000000000 + dn;
  	return !(*difference > 0);
	}

double rms_error(double m, double c) {
  	int i;
  	double mean;
  	double error_sum = 0;
  
  	for(i=0; i<n_data; i++) {
    	error_sum += residual_error(data[i].x, data[i].y, m, c);  
  	}
  
  	mean = error_sum / n_data;
  
  	return sqrt(mean);
}

__global__ void find_err(mean_intercept *mi, point_t *data, double *eacherror) {
  	int i;
  	i = threadIdx.x + blockIdx.x * blockDim.x;
	eacherror[i] = residual_err(data[i].x, data[i].y, mi->mean, mi->intercept);  	
}



int main() {
  	int i;
  	double bm = 1.3;
  	double bc = 10;
	double be;
	double dm[8];
  	double dc[8];
  	double e[8];
 	double step = 0.01;
  	double best_error = 999999999;
  	int best_error_i;
  	int minimum_found = 0;

  	struct timespec start, finish;   
  	long long int time_elapsed;
	double *eacherror;
	point_t *d_data;
	mean_intercept *mi;
	double total_error;
	double mean;

	
  	double om[] = {0,1,1, 1, 0,-1,-1,-1};
  	double oc[] = {1,1,0,-1,-1,-1, 0, 1};
	
	// allocate Unified Memory -- accessible from both CPU and GPU  
	hipMallocManaged((void **)&mi, sizeof(mean_intercept) * 8);
	hipMallocManaged(&eacherror, sizeof(double) * 1000);

	// dynamically allocates memory on device (GPU)
	hipMalloc(&d_data, sizeof(point_t) * 1000);

  	clock_gettime(CLOCK_MONOTONIC, &start);

  	be = rms_error(bm, bc);

	// Transfer data from CPU (host) to GPU (device)
	hipMemcpy(d_data,data, sizeof(data), hipMemcpyHostToDevice);

	// defining block and grid dimensions of (10(x),1(y),1(z)) and (100 (x), 1(y), 1(x))
	dim3 bd(10, 1, 1);
  	dim3 gd(100, 1, 1);

  	while(!minimum_found) {
    	for(i=0;i<8;i++) {
     		dm[i] = bm + (om[i] * step);
      		dc[i] = bc + (oc[i] * step);

			mi[i].mean = dm[i];
			mi[i].intercept = dc[i];
			//executes kernal function passing three variables as parameter
 		 	// <<<gd,bd>>> represents grid and block dimensions respectively
      		find_err<<<gd,bd>>>(&mi[i],d_data,eacherror);
			
			// Wait for GPU to finish before accessing on host
			hipDeviceSynchronize();
	
			for (int k = 0; k < 1000; k++){
				
				total_error += eacherror[k];
			}
				mean = total_error/1000 ;
							

				e[i] = sqrt(mean);
		  		if(e[i] < best_error) {
		    		best_error = e[i];
		    		best_error_i = i;
      			}
				total_error = 0;
		}
    	

    	/*printf("best m,c is %lf,%lf with error %lf in direction %d\n", 
      	dm[best_error_i], dc[best_error_i], best_error, best_error_i);*/
    	if(best_error < be) {
      		be = best_error;
      		bm = dm[best_error_i];
      		bc = dc[best_error_i];
    	} else {
      	minimum_found = 1;
    	}
  	}
  	printf("minimum m,c is %lf,%lf with error %lf\n", bm, bc, be);
	
  	clock_gettime(CLOCK_MONOTONIC, &finish);
  	time_difference(&start, &finish, &time_elapsed);
  	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,(time_elapsed/1.0e9));

	// Free memory
	hipFree(&mi);
	hipFree(&eacherror);
	hipFree(&d_data); 
  	return 0;
}


