#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o cuda_crack cuda_crack.cu
    ./cuda_crack
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ void is_a_match(char *attempt) {
	char *plain_password[] = {(char *)"AV72",(char *)"FR24",(char *)"HS31",(char *)"TD14"};
  
	for(int x = 0; x < 4; x++){
	  	char *a = attempt;
	  	char *p = plain_password[x];
	  
	  	while(*a == *p) {
			if(*a == '\0') {
		  	printf("password found: %s\n", plain_password[x]);
		  	break;
			}
			a++;
			p++;
	  	}
	}
}

/****************************************************************************
  The kernel function assume that there will be only one thread and uses 
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference) {
  	long long int ds =  finish->tv_sec - start->tv_sec; 
  	long long int dn =  finish->tv_nsec - start->tv_nsec; 

  	if(dn < 0 ) {
    	ds--;
    	dn += 1000000000; 
  	} 
 	*difference = ds * 1000000000 + dn;
  	return !(*difference > 0);
}

__global__ void  kernel() {
	char i,j,k,l;
  
  	char password[5];
	
  	password[4] = '\0';
  
	//starting value is A as it's ascii value is 65
	//initially threadIdx and blockIdx values are 0
  	i = threadIdx.x+65; 
    j = blockIdx.x+65;
	password[0] = i;
    password[1] = j;

	for (k = '0'; k<='9'; k++){
		for(l='0'; l<= '9'; l++){
			password[2] = k;
			password[3] = l;
		  	is_a_match(password);
		}
	}
}

int main() {
	struct timespec start, finish;   
  	long long int time_elapsed;

	// defining block and grid dimensions of (26(x),1(y),1(z)) and (26 (x), 1(y), 1(x))
	dim3 gd(26, 1, 1); 
  	dim3 bd(26, 1, 1);

	clock_gettime(CLOCK_MONOTONIC, &start);
	
	// <<<gd,bd>>> represents grid and block dimensions respectively
  	kernel <<<gd, bd>>>();

	//blocks until the device has completed all preceding requested task
  	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
  	time_difference(&start, &finish, &time_elapsed);
  	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
         (time_elapsed/1.0e9)); 
  	return 0;
}


